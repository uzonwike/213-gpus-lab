
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define N 32
#define THREADS_PER_BLOCK 32

__global__ void dotproduct(float* x, float* y, float* result) {
    // Compute the index this thread should use to access elements
    size_t index = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

    // Create space for a shared array that all threads in this block will
    // use to store pairwise products
    __shared__ float temp[THREADS_PER_BLOCK];

    // Compute pairwise products
    temp[threadIdx.x] = x[index] * y[index];

    __syncthreads();
    // The thread with index zero will sum up the values in temp
    if(threadIdx.x == 0) {
        float sum = 0;
        int i;
        for(i=0; i<THREADS_PER_BLOCK; i++) {
            atomicAdd(&sum, temp[i]);
        }

        // Add the sum for this block to the result
        *result += sum;
    }
}

int main() {
    // Allocate arrays for X and Y on the CPU
    float* cpu_x = (float*)malloc(sizeof(float) * N);
    float* cpu_y = (float*)malloc(sizeof(float) * N);

    // Initialize X and Y
    int i;
    for(i=0; i<N; i++) {
        cpu_x[i] = (float)i;
        cpu_y[i] = (float)i;
    }

    // Allocate space for X and Y on the GPU
    float* gpu_x;
    float* gpu_y;
    float gpu_result = 0.0;

    if(hipMalloc(&gpu_x, sizeof(float) * N) != hipSuccess) {
        fprintf(stderr, "Failed to allocate X array on GPU\n");
        exit(2);
    }

    if(hipMalloc(&gpu_y, sizeof(float) * N) != hipSuccess) {
        fprintf(stderr, "Failed to allocate Y array on GPU\n");
        exit(2);
    }

    // Copy the host X and Y arrays to the device X and Y
    // arrays
    if(hipMemcpy(gpu_x, cpu_x, sizeof(float) * N, hipMemcpyHostToDevice) !=
            hipSuccess) {
        fprintf(stderr, "Failed to copy X to the GPU\n");
    }

    if(hipMemcpy(gpu_y, cpu_y, sizeof(float) * N,
                hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy Y to the GPU\n");
    }

    // How many blocks should be run, rounding up to
    // include all threads?
    size_t blocks = (N + THREADS_PER_BLOCK - 1) /
        THREADS_PER_BLOCK;

    // Run the saxpy kernel
    dotproduct<<<blocks, THREADS_PER_BLOCK>>>(gpu_x, gpu_y, &gpu_result);

    // Wait for the kernel to finish
    hipDeviceSynchronize();


    printf("%f\n", gpu_result);

    hipFree(gpu_x);
    hipFree(gpu_y);
    free(cpu_x);
    free(cpu_y);

    return 0;
}
